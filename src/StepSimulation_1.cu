#include "hip/hip_runtime.h"

#pragma once
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <numeric>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
#include "kernel.cuh"
#include <map>
#include<tuple>
#include <algorithm>
#include <random>
/*
LINGYIN��
ȫ�ֱ���˵����
glm::vec3* dev_pos: λ�����飨GPU��
glm::vec3* dev_vel1: �ٶ�����1��GPU��
glm::vec3* dev_vel2: �ٶ�����2��GPU��
int n: boids ����

����Ҫ�����ǣ�
���ݵ�ǰ���ٶȼ����µ��ٶȣ�Ȼ������µ��ٶȸ���λ�á�
����� dt ����һ��ʱ�䲽�������ڸ���λ��ʱλ�Ƶ�Ȩ�� 
pos = pos + new_speed * dt��
*/


void Boids::StepSimulation_1(float dt) {
    static int __ = 0;
    int n = numObjects;
    const int Block = 512;

    static glm::vec3* dev_org_speed = nullptr;
    static glm::vec3* dev_new_speed = nullptr;
    static int* dev_index = nullptr;

    int used_count_of_thread = (n + 512 - 1) / 512 * 512;
    __++;
    if (__ == 1) {
        std::vector<int> index(used_count_of_thread);
        hipMalloc((void**)&dev_index, used_count_of_thread * sizeof(int));
        for (int i = 0; i < used_count_of_thread; i++) {
            index[i] = std::min(i, n - 1);
        }
        hipMemcpy(dev_index, index.data(), used_count_of_thread * sizeof(int), hipMemcpyHostToDevice);

        dev_org_speed = dev_vel1;
        dev_new_speed = dev_vel2;
        CUDA_CHECK_KERNEL();
    }
    std::swap(dev_org_speed, dev_new_speed);


    hipDeviceSynchronize();

    CalculateNewSpeedNaive << <used_count_of_thread / Block, Block >> > (
        n, dev_pos, dev_org_speed, dev_new_speed, dev_index);
    hipDeviceSynchronize();
    CUDA_CHECK_KERNEL();

    MoveBoids << <n / Block, Block >> > (dt, dev_pos, dev_new_speed);
    CUDA_CHECK_KERNEL();
}