#include "hip/hip_runtime.h"

#pragma once
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <numeric>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
#include "kernel.cuh"
#include <map>
#include<tuple>
#include <algorithm>
#include <random>

void Boids::StepSimulation_3(float dt) {

    static int __ = 0;

    static int* dev_box_index = nullptr;
    static int* dev_index = nullptr;
    static int* dev_launch_index = nullptr;
    static int* dev_begin_of_box = nullptr;
    static int* dev_size_of_box = nullptr;

    static char* dev_buffer = nullptr;
    static char* host_buffer = nullptr;
    static glm::vec3* dev_org_speed = nullptr;
    static glm::vec3* dev_new_speed = nullptr;

    int n = numObjects;
    const int B = 32;
    const int BBB = B * B * B;
    const int max_count_of_thread = BBB + n;

    const int size_of_buffer = (
        n * sizeof(int) +
        n * sizeof(int) +
        max_count_of_thread * sizeof(int) +
        BBB * sizeof(int) +
        BBB * sizeof(int));
    //const int size_of_pos = n * sizeof(glm::vec3);

    static int* host_box_index = nullptr;
    static int* host_index = nullptr;
    static int* host_launch_index = nullptr;
    static int* host_begin_of_box = nullptr;
    static int* host_size_of_box = nullptr;
    static std::vector<glm::vec3> host_pos;

    __++;
    if (__ == 1) {
        hipMalloc((void**)&dev_buffer, size_of_buffer);
        host_buffer = new char[size_of_buffer];

        dev_box_index = (int*)(dev_buffer);
        dev_index = (int*)(dev_box_index + n);
        dev_launch_index = (int*)(dev_index + n);
        dev_begin_of_box = (int*)(dev_launch_index + max_count_of_thread);
        dev_size_of_box = (int*)(dev_begin_of_box + BBB);

        host_box_index = (int*)(host_buffer);
        host_index = (int*)(host_box_index + n);
        host_launch_index = (int*)(host_index + n);
        host_begin_of_box = (int*)(host_launch_index + max_count_of_thread);
        host_size_of_box = (int*)(host_begin_of_box + BBB);

        host_pos.resize(n);
        dev_org_speed = dev_vel1;
        dev_new_speed = dev_vel2;
        CUDA_CHECK_KERNEL();
    }

    std::swap(dev_org_speed, dev_new_speed);
    std::memset(host_buffer, 0, size_of_buffer);
    hipMemcpy(host_pos.data(), dev_pos, n * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    {
        std::vector<int > index(n);
        for (int i = 0; i < n; i++) {
            index[i] = i;
        }

        float r = rule1Distance;
        { // x
            std::sort(index.begin(), index.end(), [&](const int& a, const int& b) {
                return host_pos[a].x < host_pos[b].x;
                });
            float last_x = -FLT_MAX, count = 0, p = -1;
            for (int i = 1; i <= B; i++) {
                while (p + 1 < n && (host_pos[index[p + 1]].x <= last_x + r || count < n / B + 1)) {
                    p++;
                    count++;
                    host_box_index[index[p]] += i - 1;
                }

                last_x = host_pos[index[p]].x;
                count = 0;
            }
        }

        { // y
            std::sort(index.begin(), index.end(), [&](const int& a, const int& b) {
                return host_pos[a].y < host_pos[b].y;
                });
            float last_y = -FLT_MAX, count = 0, p = -1;
            for (int i = 1; i <= B; i++) {
                while (p + 1 < n && (host_pos[index[p + 1]].y <= last_y + r || count < n / B + 1)) {
                    p++;
                    count++;
                    host_box_index[index[p]] += (i - 1) * B;
                }
                last_y = host_pos[index[p]].y;
                count = 0;
            }
        }

        { // z
            std::sort(index.begin(), index.end(), [&](const int& a, const int& b) {
                return host_pos[a].z < host_pos[b].z;
                });
            float last_z = -FLT_MAX, count = 0, p = -1;
            for (int i = 1; i <= B; i++) {
                while (p + 1 < n && (host_pos[index[p + 1]].z <= last_z + r || count < n / B + 1)) {
                    p++;
                    count++;
                    host_box_index[index[p]] += (i - 1) * B * B;
                }
                last_z = host_pos[index[p]].z;
                count = 0;
            }
        }
    }

    for (int i = 0; i < n; i++) {
        host_index[i] = i;
    }

    std::sort(host_index, host_index + n, [&](const int& a, const int& b) {
        return host_box_index[a] < host_box_index[b];
        });

    for (int i = 0; i < n; i++) {
        if (host_box_index[i] < BBB) {
            host_size_of_box[host_box_index[i]]++;
        }
    }
    for (int i = 1; i < BBB; i++) {
        host_begin_of_box[i] = host_begin_of_box[i - 1] + host_size_of_box[i - 1];
    }

    int used_count_of_thread = 0;
    const int Block = 512;

    used_count_of_thread = (n + Block - 1) / Block * Block;
    for (int i = 0; i < used_count_of_thread; i++) {
        host_launch_index[i] = std::min(i, n - 1);
    }

    hipMemcpy(dev_buffer, host_buffer, size_of_buffer, hipMemcpyHostToDevice);
    CUDA_CHECK_KERNEL();
    hipDeviceSynchronize();

    std::cout << "used_count_of_thread: " << used_count_of_thread << std::endl;
    CalculateNewSpeed << <(n + Block - 1) / Block, Block >> > (
        n, dt, 
        dev_pos, dev_org_speed, dev_new_speed, dev_box_index,
        dev_index,
        dev_begin_of_box, dev_size_of_box, B);

}
